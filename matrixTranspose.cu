
#include <hip/hip_runtime.h>
#include<stdio.h>
#define K 32 //tile size is KxK
#define N 1024 //matrix size is NxN
__global__ void transpose_serial(int *in, int *out)
{
 for(int i=0;i<N;i++){
 for(int j=0;j<N;j++){
   out[i*N+j]=in[j*N+i];
 }
}
}
__global__ void transpose_per_row(int *in, int *out)
{
 int i=threadIdx.x;
 for(int j=0;j<N;j++){
   out[i*N+j]=in[j*N+i];
 }
}
__global__ void transpose_per_element(int *in, int *out)
{
 int i=blockIdx.x*K+threadIdx.x;
 int j=blockIdx.y*K+threadIdx.y;
 out[i*N+j]=in[j*N+i];
}
__global__ void transpose_per_element_tiled(int *in, int *out)
{
 int in_i=blockIdx.x*K; //Corner point to start reading
 int in_j=blockIdx.y*K;
 int out_i=blockIdx.y*K; //Corner point to start writing
 int out_j=blockIdx.x*K;
 int x=threadIdx.x;
 int y=threadIdx.y;
 __shared__ int tile[K][K];
 tile[y][x] = in[(in_i+x)+(in_j+y)*N];
 __syncthreads();
 out[(out_i+x)+(out_j+y)*N] = tile[x][y];
}
int main(void)
{
 int *in,*out;
 int *d_in,*d_out;
 int size = sizeof(int) * N * N;
in= (int *)(malloc(size));
 out= (int *)(malloc(size));
 //printf("Elements of in \n");
 for(int i=0;i<N;i++)
 {
  for(int j=0;j<N;j++)
  {
    in[i+j*N]=j;
  }
 }
 hipEvent_t start, stop;
 hipEventCreate(&start);
 hipEventCreate(&stop);
 hipMalloc((void **)&d_in,size);
 hipMalloc((void **)&d_out,size);
 hipMemcpy(d_in,in,size,hipMemcpyHostToDevice);
 dim3 block(N/K,N/K);
 dim3 thread(K,K);
 hipEventRecord(start);
//transpose_per_element_tiled<<<block,thread>>>(d_in,d_out);
 transpose_per_element_tiled<<<block,thread>>>(d_in,d_out);
//tranpose_per_row<<<1,N>>>
//transpose_serial<<<1,1>>>
 hipEventRecord(stop);
 hipEventSynchronize(stop);
 float ms;
 hipEventElapsedTime(&ms,start,stop);
 hipMemcpy(out,d_out,size,hipMemcpyDeviceToHost);
 //printf("Elements of out\n");
// for(int i=0;i<N;i++)
 //{
  //for(int j=0;j<N;j++)
 // {
   // printf("%d\t",out[i+j*N]);
 // }
// }
 printf("Time taken = %f \n",ms);
 hipFree(d_in);
 hipFree(d_out);
 return 0;
}
