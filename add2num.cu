
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void kernel(int * a, int * b)
{
 *b=*a+*b;
}
int main(void)
{
int h_in,h_out;
int *d_out,*d_in;
h_in=2;
h_out=7;
hipMalloc((void **)&d_out,sizeof(int));
hipMalloc((void **)&d_in,sizeof(int));
hipMemcpy(d_in,&h_in,sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_out,&h_out,sizeof(int),hipMemcpyHostToDevice);
kernel<<<1,1>>>(d_in,d_out);
hipMemcpy(&h_out,d_out,sizeof(int),hipMemcpyDeviceToHost);
printf("%d\n",h_out);
hipFree(d_in);
hipFree(d_out);
return 0;
}
