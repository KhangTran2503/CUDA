
#include <hip/hip_runtime.h>
#include<stdio.h>
#define BLOCK_DIM 25
#define N 25
__global__ void matadd(int *a, int *b, int *c)
{
 int col=blockIdx.x*blockDim.x+threadIdx.x;
 int row=blockIdx.y*blockDim.y+threadIdx.y;
 int index = col + row*N;
 if(col<N && row<N){
   c[index]=a[index]+b[index];
 }
}
int main(void)
{
 int a[N][N],b[N][N],c[N][N];
 int *d_a,*d_b,*d_c;
 int size = sizeof(int) * N * N;
 printf("Elements of matA\n");
 for(int i=0;i<N;i++)
 {
  for(int j=0;j<N;j++)
  {
    a[i][j]=i;
    printf("%d\t",a[i][j]);
  }
  printf("\n");
 }
 printf("Elements of matB\n");
 for(int i=0;i<N;i++)
 {
  for(int j=0;j<N;j++)
  {
    b[i][j]=j;
    printf("%d\t",b[i][j]);
  }
  printf("\n");
 }
 printf("\n");
 hipMalloc((void **)&d_a,size);
 hipMalloc((void **)&d_b,size);
 hipMalloc((void **)&d_c,size);
 hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
 hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
 dim3 dimBlock(BLOCK_DIM,BLOCK_DIM);
 dim3 dimGrid((int)(N/dimBlock.x),(int)(N/dimBlock.y));
 matadd<<<dimGrid,dimBlock>>>(d_a,d_b,d_c);
 hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
 printf("Elements of MatC\n");
 for(int i=0;i<N;i++)
 {
  for(int j=0;j<N;j++)
  {
    printf("%d\t",c[i][j]);
  }
  printf("\n");
 }
 hipFree(d_a);
 hipFree(d_b);
 hipFree(d_c);
 return 0;
}
