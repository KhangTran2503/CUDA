
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void add(int *a, int *b, int *c)
{
 int id = threadIdx.x;
 c[id]=a[id]+b[id];
}
int main(void)
{
 const int a[5] = {1,2,3,4,5};
 const int b[5] = {10,20,30,40,50};
 int c[5];
 int *d_a,*d_b,*d_c;
 int size = sizeof(int)*5;
 hipMalloc((void **)&d_a,sizeof(int)*5);
 hipMalloc((void **)&d_b,sizeof(int)*5);
 hipMalloc((void **)&d_c,sizeof(int)*5);
 hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
 hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
 add<<<1,5>>>(d_a,d_b,d_c);
 hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
 for(int i=0;i<5;i++)
 {
        printf("%d\t",c[i]);
 }
 hipFree(d_a);
 hipFree(d_b);
 hipFree(d_c);
 return 0;
}
