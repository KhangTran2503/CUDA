

#include <hip/hip_runtime.h>
#include<stdio.h>
#define BLOCK_DIM 1
#define N 16
__global__ void matmul(int *a, int *b, int *c, int width)
{
 int k, sum=0;
 int col=blockIdx.x*blockDim.x+threadIdx.x;
 int row=blockIdx.y*blockDim.y+threadIdx.y;
 if(col<width && row<width){
  for(k=0;k<width;k++){
   sum+=a[row*width+k]*b[k*width+col];
   c[row*width+col]=sum;
  }
 }
}
int main(void)
{
 int a[N][N],b[N][N],c[N][N];
 int *d_a,*d_b,*d_c;
 int size = sizeof(int) * N * N;
 printf("Elements of matA\n");
 for(int i=0;i<N;i++)
 {
  for(int j=0;j<N;j++)
  {
    a[i][j]=i;
    printf("%d\t",a[i][j]);
  }
  printf("\n");
 }
 printf("Elements of matB\n");
 for(int i=0;i<N;i++)
 {
  for(int j=0;j<N;j++)
  {
    b[i][j]=j;
    printf("%d\t",b[i][j]);
  }
  printf("\n");
 }
 printf("\n");
 hipMalloc((void **)&d_a,size);
 hipMalloc((void **)&d_b,size);
 hipMalloc((void **)&d_c,size);
 hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
 hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
 dim3 dimBlock(BLOCK_DIM,BLOCK_DIM);
 dim3 dimGrid((int)(N/dimBlock.x),(int)(N/dimBlock.y));
 matmul<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,N);
 hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
 printf("Elements of MatC\n");
 for(int i=0;i<N;i++)
 {
  for(int j=0;j<N;j++)
  {
    printf("%d\t",c[i][j]);
  }
  printf("\n");
 }
 hipFree(d_a);
 hipFree(d_b);
 hipFree(d_c);
 return 0;
}
