
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void cube(int * a, int * b)
{
 int id=blockIdx.x*blockDim.x+threadIdx.x;
 b[id]=a[id]*a[id]*a[id];
}
#define N 25
#define B 5
int main(void)
{
int a[N],b[N];
int *d_a,*d_b;
for(int i=0;i<N;i++)
{
  a[i]=int(i);
}
hipMalloc((void **)&d_a,N*sizeof(int));
hipMalloc((void **)&d_b,N*sizeof(int));
hipMemcpy(d_a,a,N*sizeof(int),hipMemcpyHostToDevice);
cube<<<N/B,B>>>(d_a,d_b);
hipMemcpy(b,d_b,N*sizeof(int),hipMemcpyDeviceToHost);
for(int i=0;i<N;i++)
{
printf("Cube of %d = %d\n",a[i],b[i]);
}
hipFree(d_a);
hipFree(d_b);
return 0;
}
